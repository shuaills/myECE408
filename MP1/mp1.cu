#include "hip/hip_runtime.h"
// MP 1
#include "../libwb/wb.h"

__global__ void vecAdd(float *in1, float *in2, float *out, int len) {
  //@@ Insert code to implement vector addition here
  const int n = blockDim.x * blockIdx.x + threadIdx.x;
  if(n < len) out[n] = in1[n] + in2[n]; 
}

int main(int argc, char **argv) {
  wbArg_t args;
  int inputLength;
  float *hostInput1;
  float *hostInput2;
  float *hostOutput;
  float *deviceInput1;
  float *deviceInput2;
  float *deviceOutput;

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostInput1 =
      (float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
  if (!hostInput1) {
    printf("Error: wbImport failed for input file 1\n");
    return -1;
  }

  hostInput2 =
      (float *)wbImport(wbArg_getInputFile(args, 1), &inputLength);
  hostOutput = (float *)malloc(inputLength * sizeof(float));
  printf("inputLength: %d\n", inputLength);
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The input length is ", inputLength);

  wbTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here
  hipMalloc((void **) &deviceInput1, inputLength * sizeof(float));
  hipMalloc((void **) &deviceInput2, inputLength * sizeof(float));
  hipMalloc((void **) &deviceOutput, inputLength * sizeof(float));
  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here
  hipMemcpy(deviceInput1, hostInput1, inputLength * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(deviceInput2, hostInput2, inputLength * sizeof(float), hipMemcpyHostToDevice);

  wbTime_stop(GPU, "Copying input memory to the GPU.");

  //@@ Initialize the grid and block dimensions here
  dim3 grid(ceil((float)inputLength / 256), 1, 1);
  dim3 block(256, 1, 1);

  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Launch the GPU Kernel here
  vecAdd<<<grid, block>>>(deviceInput1, deviceInput2, deviceOutput, inputLength);
  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostOutput, deviceOutput, inputLength * sizeof(float), hipMemcpyDeviceToHost);
  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here
  hipFree(deviceInput1);
  hipFree(deviceInput2);
  hipFree(deviceOutput);

  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostOutput, inputLength);

  free(hostInput1);
  free(hostInput2);
  free(hostOutput);

  return 0;
}
